#include <torch/library.h>
#include <ATen/ATen.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/CUDAUtils.h>
#include <ATen/cuda/HIPContext.h>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/epilogue/threadblock/fusion/visitors.hpp"
#include "cutlass/gemm/kernel/default_gemm_universal_with_visitor.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"


#define CUTLASS_CHECK(status) \
  TORCH_CHECK(status == cutlass::Status::kSuccess, "cutlass error: ", cutlassGetStatusString(status))


// define common params
using ElementInput       = cutlass::float_e4m3_t;
using ElementOutput      = cutlass::bfloat16_t;
using ElementScale       = float;
using ElementAccumulator = float;
using OpClass            = cutlass::arch::OpClassTensorOp;
using ArchTag            = cutlass::arch::Sm89;

constexpr int AlignmentOutput = 128 / cutlass::sizeof_bits<ElementOutput>::value;

at::Tensor cutlass_sm89_fp8_mm(at::Tensor A, at::Tensor B) {
  int M = A.size(0);
  int K = A.size(1);
  int N = B.size(1);
  at::Tensor C = at::empty({M, N}, A.options().dtype(at::kBFloat16));

  // TODO: use better config
  // static int const kStages = 3;
  using Gemm = cutlass::gemm::device::Gemm<
    ElementInput, cutlass::layout::RowMajor,    // A matrix
    ElementInput, cutlass::layout::ColumnMajor, // B matrix
    ElementOutput, cutlass::layout::RowMajor,   // C matrix
    ElementAccumulator, OpClass, ArchTag
  >;
  Gemm::Arguments args {
    {M, N, K},
    {reinterpret_cast<ElementInput *>(A.data_ptr()), K},
    {reinterpret_cast<ElementInput *>(B.data_ptr()), K},
    {reinterpret_cast<ElementOutput *>(C.data_ptr()), N},
    {reinterpret_cast<ElementOutput *>(C.data_ptr()), N},
    {1, 0}  // epilogue
  };
  Gemm gemm_op;
  auto stream = at::cuda::getCurrentCUDAStream();
  CUTLASS_CHECK(gemm_op(args, nullptr, stream));

  return C;
}

// this function is based on the following cutlass example
// https://github.com/NVIDIA/cutlass/blob/main/examples/47_ampere_gemm_universal_streamk/ampere_gemm_universal_streamk_broadcast.cu
// also with the help of emitted code from cutlass Python
at::Tensor cutlass_sm89_row_scaled_fp8_mm(at::Tensor A, at::Tensor B, at::Tensor scale_A, at::Tensor scale_B) {
  int M = A.size(0);
  int K = A.size(1);
  int N = B.size(1);
  at::Tensor out = at::empty({M, N}, A.options().dtype(at::kBFloat16));

  const ElementInput *A_ptr       = reinterpret_cast<ElementInput *>(A.data_ptr());
  const ElementInput *B_ptr       = reinterpret_cast<ElementInput *>(B.data_ptr());
  const ElementScale *scale_A_ptr = reinterpret_cast<ElementScale *>(scale_A.data_ptr());
  const ElementScale *scale_B_ptr = reinterpret_cast<ElementScale *>(scale_B.data_ptr());
  ElementOutput *out_ptr          = reinterpret_cast<ElementOutput *>(out.data_ptr());

  // https://github.com/NVIDIA/cutlass/blob/v3.9.2/examples/58_ada_fp8_gemm/ada_fp8_gemm.cu
  using ThreadblockShape = cutlass::gemm::GemmShape<128, 64, 128>;
  using WarpShape        = cutlass::gemm::GemmShape<64, 32, 128>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 32>;
  constexpr int numStages = 3;

  // build epilogue visitor tree
  constexpr auto RoundMode = cutlass::FloatRoundStyle::round_to_nearest;
  constexpr int numEpilogueStages = 1;

  using namespace cute;
  using namespace cutlass::epilogue::threadblock;
  using ElementEpilogue = float;
  using Multiply = VisitorCompute<cutlass::multiplies, ElementEpilogue, ElementEpilogue, RoundMode>;
  using OutputTileThreadMap = OutputTileThreadLayout<ThreadblockShape, WarpShape, ElementOutput, AlignmentOutput, numEpilogueStages>;

  // (1, N). stride is MNL (whatever that is)
  using ColScale = VisitorRowBroadcast<OutputTileThreadMap, ElementScale, Stride<_0, _1, int32_t>>;
  using EVTCompute0 = Sm80EVT<Multiply, VisitorAccFetch, ColScale>;

  // (M, 1)
  using RowScale = VisitorColBroadcast<OutputTileThreadMap, ElementScale, Stride<_1, _0, int32_t>>;
  using EVTCompute1 = Sm80EVT<Multiply, EVTCompute0, RowScale>;

  using Output = VisitorAuxStore<OutputTileThreadMap, ElementOutput, RoundMode, Stride<int64_t, _1, int64_t>>;
  using EVTOutput = Sm80EVT<Output, EVTCompute1>;

  constexpr int AlignmentInput = 128 / cutlass::sizeof_bits<ElementInput>::value;
  using EVTKernel = typename cutlass::gemm::kernel::DefaultGemmWithVisitor<
    ElementInput, cutlass::layout::RowMajor,    cutlass::ComplexTransform::kNone, AlignmentInput,
    ElementInput, cutlass::layout::ColumnMajor, cutlass::ComplexTransform::kNone, AlignmentInput,
    ElementOutput, cutlass::layout::RowMajor, AlignmentOutput,
    ElementAccumulator, ElementEpilogue, OpClass, ArchTag,
    ThreadblockShape, WarpShape, InstructionShape,
    EVTOutput,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<1>,
    numStages,
    cutlass::arch::OpMultiplyAdd,
    numEpilogueStages
  >::GemmKernel;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<EVTKernel>;

  typename EVTOutput::Arguments callback_args{
    {
      {
        {},                                                        // Accum
        {scale_B_ptr, ElementScale(0), {_0{}, _1{}, int32_t(N)}},  // ColScale
        {}                                                         // Multiply
      },                                                           // EVTCompute0
      {scale_A_ptr, ElementScale(0), {_1{}, _0{}, int32_t(M)}},    // RowScale
      {}                                                           // Multiply
    },                                                             // EVTCompute1
    {out_ptr, {int64_t{N}, _1{}, int64_t{M*N}}}                    // EVTOutput
  };

  typename Gemm::Arguments args(
    cutlass::gemm::GemmUniversalMode::kGemm,
    cutlass::gemm::GemmCoord{M, N, K},
    1,                              // batch_split
    callback_args,
    A_ptr, B_ptr, nullptr, nullptr, // unsued C_ptr and D_ptr
    M * K, N * K, 0, 0,             // batch_stride A, B, C, D
    K, K, 0, 0                      // stride A, B, C, D
  );

  Gemm gemm;
  CUTLASS_CHECK(gemm.can_implement(args));

  auto stream = at::cuda::getCurrentCUDAStream();
  CUTLASS_CHECK(gemm(args, nullptr, stream));

  return out;
}

TORCH_LIBRARY_IMPL(gn_kernels, CUDA, m) {
  m.impl("gn_kernels::cutlass_sm89_fp8_mm", &cutlass_sm89_fp8_mm);
  m.impl("gn_kernels::cutlass_sm89_row_scaled_fp8_mm", &cutlass_sm89_row_scaled_fp8_mm);
}
